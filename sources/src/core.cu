#include "hip/hip_runtime.h"
#include "core.h"

namespace v0 //cuda baseline
{
    static __global__ void cudaCallbackKernel(
        const int width,
        const int height,
        const float *__restrict__ input,
        float *__restrict__ output)
    {
        const int idy = blockIdx.y * blockDim.y + threadIdx.y;
        const int idx = blockIdx.x * blockDim.x + threadIdx.x;
        if (idy < height && idx < width)
        {
            int cnt[16] = {0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0};
            for (int offsety = -2; offsety <= 2; ++offsety)
            {
                const int py = idy + offsety;
                if (0 <= py && py < height)
                    for (int offsetx = -2; offsetx <= 2; ++offsetx)
                    {
                        const int px = idx + offsetx;
                        if (0 <= px && px < width)
                            ++cnt[(int)input[py * width + px]];
                    }
            }
            double
                n = (min(idx, 2) + 1 + min(width - idx, 2)) * (min(idy, 2) + 1 + min(height - idy, 2)),
                n_inv = 1.0 / n,
                ans = log(n);
            for (int i = 0; i < 16; ++i)
                if (cnt[i])
                    ans -= log((double)cnt[i]) * cnt[i] * n_inv;
            output[idy * width + idx] = ans;
        }
    }

    static void cudaCallback(
        int width,
        int height,
        float *sample,
        float **result)
    {
        float *input_d, *output_d;

        CHECK(hipMalloc((void **)&input_d, sizeof(float) * width * height));
        CHECK(hipMalloc((void **)&output_d, sizeof(float) * width * height));
        CHECK(hipMemcpy(input_d, sample, sizeof(float) * width * height, hipMemcpyHostToDevice));

        const int
            BLOCK_DIM_X = 32,
            BLOCK_DIM_Y = 32;

        const dim3
            blockDim(BLOCK_DIM_X, BLOCK_DIM_Y),
            gridDim(divup(width, BLOCK_DIM_X), divup(height, BLOCK_DIM_Y));

        cudaCallbackKernel<<<
            gridDim,
            blockDim>>>(
            width,
            height,
            input_d,
            output_d);

        *result = (float *)malloc(sizeof(float) * width * height);
        CHECK(hipMemcpy(*result, output_d, sizeof(float) * width * height, hipMemcpyDeviceToHost));
        CHECK(hipFree(input_d));
        CHECK(hipFree(output_d));
    }
} // namespace v0
namespace v1 //cuda 预处理log到寄存器
{
    static __global__ void cudaCallbackKernel(
        const int width,
        const int height,
        const float *__restrict__ input,
        float *__restrict__ output)
    {
        const int idy = blockIdx.y * blockDim.y + threadIdx.y;
        const int idx = blockIdx.x * blockDim.x + threadIdx.x;
        if (idy < height && idx < width)
        {
            int cnt[16] = {0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0};
            for (int offsety = -2; offsety <= 2; ++offsety)
            {
                const int py = idy + offsety;
                if (0 <= py && py < height)
                    for (int offsetx = -2; offsetx <= 2; ++offsetx)
                    {
                        const int px = idx + offsetx;
                        if (0 <= px && px < width)
                            ++cnt[(int)input[py * width + px]];
                    }
            }
            const double mylog[26] = {
                0.0,
                log(1.0),
                log(2.0),
                log(3.0),
                log(4.0),
                log(5.0),
                log(6.0),
                log(7.0),
                log(8.0),
                log(9.0),
                log(10.0),
                log(11.0),
                log(12.0),
                log(13.0),
                log(14.0),
                log(15.0),
                log(16.0),
                log(17.0),
                log(18.0),
                log(19.0),
                log(20.0),
                log(21.0),
                log(22.0),
                log(23.0),
                log(24.0),
                log(25.0)};

            const int n = (min(idx, 2) + 1 + min(width - idx, 2)) * (min(idy, 2) + 1 + min(height - idy, 2));
            double ans = mylog[n], n_inv = 1.0 / n;
            for (int i = 0; i < 16; ++i)
                ans -= mylog[cnt[i]] * cnt[i] * n_inv;
            output[idy * width + idx] = ans;
        }
    }

    static void cudaCallback(
        int width,
        int height,
        float *sample,
        float **result)
    {
        float *input_d, *output_d;

        CHECK(hipMalloc((void **)&input_d, sizeof(float) * width * height));
        CHECK(hipMalloc((void **)&output_d, sizeof(float) * width * height));
        CHECK(hipMemcpy(input_d, sample, sizeof(float) * width * height, hipMemcpyHostToDevice));

        const int
            BLOCK_DIM_X = 32,
            BLOCK_DIM_Y = 32;

        const dim3
            blockDim(BLOCK_DIM_X, BLOCK_DIM_Y),
            gridDim(divup(width, BLOCK_DIM_X), divup(height, BLOCK_DIM_Y));

        cudaCallbackKernel<<<
            gridDim,
            blockDim>>>(
            width,
            height,
            input_d,
            output_d);

        *result = (float *)malloc(sizeof(float) * width * height);
        CHECK(hipMemcpy(*result, output_d, sizeof(float) * width * height, hipMemcpyDeviceToHost));
        CHECK(hipFree(input_d));
        CHECK(hipFree(output_d));
    }
} // namespace v1
namespace v2 //cuda 预处理log到shared memory
{
    static __global__ void cudaCallbackKernel(
        const int width,
        const int height,
        const float *__restrict__ input,
        float *__restrict__ output)
    {
        const int idy = blockIdx.y * blockDim.y + threadIdx.y;
        const int idx = blockIdx.x * blockDim.x + threadIdx.x;

        __shared__ double mylog[26];
        if (threadIdx.y == 0 && threadIdx.x < 26)
            mylog[threadIdx.x] = threadIdx.x == 0 ? 0.0 : log((double)threadIdx.x);
        __syncthreads();
        if (idy < height && idx < width)
        {
            int cnt[16] = {0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0};
            for (int offsety = -2; offsety <= 2; ++offsety)
            {
                const int py = idy + offsety;
                if (0 <= py && py < height)
                    for (int offsetx = -2; offsetx <= 2; ++offsetx)
                    {
                        const int px = idx + offsetx;
                        if (0 <= px && px < width)
                            ++cnt[(int)input[py * width + px]];
                    }
            }
            const int n = (min(idx, 2) + 1 + min(width - idx, 2)) * (min(idy, 2) + 1 + min(height - idy, 2));
            double ans = mylog[n], n_inv = 1.0 / n;
            for (int i = 0; i < 16; ++i)
                ans -= mylog[cnt[i]] * cnt[i] * n_inv;
            output[idy * width + idx] = ans;
        }
    }

    static void cudaCallback(
        int width,
        int height,
        float *sample,
        float **result)
    {
        float *input_d, *output_d;

        CHECK(hipMalloc((void **)&input_d, sizeof(float) * width * height));
        CHECK(hipMalloc((void **)&output_d, sizeof(float) * width * height));
        CHECK(hipMemcpy(input_d, sample, sizeof(float) * width * height, hipMemcpyHostToDevice));

        const int
            BLOCK_DIM_X = 32,
            BLOCK_DIM_Y = 32;

        const dim3
            blockDim(BLOCK_DIM_X, BLOCK_DIM_Y),
            gridDim(divup(width, BLOCK_DIM_X), divup(height, BLOCK_DIM_Y));

        cudaCallbackKernel<<<
            gridDim,
            blockDim>>>(
            width,
            height,
            input_d,
            output_d);

        *result = (float *)malloc(sizeof(float) * width * height);
        CHECK(hipMemcpy(*result, output_d, sizeof(float) * width * height, hipMemcpyDeviceToHost));
        CHECK(hipFree(input_d));
        CHECK(hipFree(output_d));
    }
} // namespace v2
namespace v3 //cuda 预处理log到constant memory
{
    static __constant__ double mylog[26];
    static struct InitPlogp
    {
        InitPlogp()
        {
            const double mylog_h[26] = {
                0.0,
                log(1.0),
                log(2.0),
                log(3.0),
                log(4.0),
                log(5.0),
                log(6.0),
                log(7.0),
                log(8.0),
                log(9.0),
                log(10.0),
                log(11.0),
                log(12.0),
                log(13.0),
                log(14.0),
                log(15.0),
                log(16.0),
                log(17.0),
                log(18.0),
                log(19.0),
                log(20.0),
                log(21.0),
                log(22.0),
                log(23.0),
                log(24.0),
                log(25.0)};
            hipMemcpyToSymbol(HIP_SYMBOL(mylog), mylog_h, sizeof(double) * 26);
        }
    } tmpInit;

    static __global__ void cudaCallbackKernel(
        const int width,
        const int height,
        const float *__restrict__ input,
        float *__restrict__ output)
    {
        const int idy = blockIdx.y * blockDim.y + threadIdx.y;
        const int idx = blockIdx.x * blockDim.x + threadIdx.x;
        if (idy < height && idx < width)
        {
            int cnt[16] = {0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0};
            for (int offsety = -2; offsety <= 2; ++offsety)
            {
                const int py = idy + offsety;
                if (0 <= py && py < height)
                    for (int offsetx = -2; offsetx <= 2; ++offsetx)
                    {
                        const int px = idx + offsetx;
                        if (0 <= px && px < width)
                            ++cnt[(int)input[py * width + px]];
                    }
            }
            const int n = (min(idx, 2) + 1 + min(width - idx, 2)) * (min(idy, 2) + 1 + min(height - idy, 2));
            double ans = mylog[n], n_inv = 1.0 / n;
            for (int i = 0; i < 16; ++i)
                ans -= mylog[cnt[i]] * cnt[i] * n_inv;
            output[idy * width + idx] = ans;
        }
    }

    static void cudaCallback(
        int width,
        int height,
        float *sample,
        float **result)
    {
        float *input_d, *output_d;

        CHECK(hipMalloc((void **)&input_d, sizeof(float) * width * height));
        CHECK(hipMalloc((void **)&output_d, sizeof(float) * width * height));
        CHECK(hipMemcpy(input_d, sample, sizeof(float) * width * height, hipMemcpyHostToDevice));

        const int
            BLOCK_DIM_X = 32,
            BLOCK_DIM_Y = 32;

        const dim3
            blockDim(BLOCK_DIM_X, BLOCK_DIM_Y),
            gridDim(divup(width, BLOCK_DIM_X), divup(height, BLOCK_DIM_Y));

        cudaCallbackKernel<<<
            gridDim,
            blockDim>>>(
            width,
            height,
            input_d,
            output_d);

        *result = (float *)malloc(sizeof(float) * width * height);
        CHECK(hipMemcpy(*result, output_d, sizeof(float) * width * height, hipMemcpyDeviceToHost));
        CHECK(hipFree(input_d));
        CHECK(hipFree(output_d));
    }
} // namespace v3
namespace v4 //cuda 预处理log到device memory
{
    static __device__ double mylog[26];
    static struct InitPlogp
    {
        InitPlogp()
        {
            const double mylog_h[26] = {
                0.0,
                log(1.0),
                log(2.0),
                log(3.0),
                log(4.0),
                log(5.0),
                log(6.0),
                log(7.0),
                log(8.0),
                log(9.0),
                log(10.0),
                log(11.0),
                log(12.0),
                log(13.0),
                log(14.0),
                log(15.0),
                log(16.0),
                log(17.0),
                log(18.0),
                log(19.0),
                log(20.0),
                log(21.0),
                log(22.0),
                log(23.0),
                log(24.0),
                log(25.0)};
            hipMemcpyToSymbol(HIP_SYMBOL(mylog), mylog_h, sizeof(double) * 26);
        }
    } tmpInit;

    static __global__ void cudaCallbackKernel(
        const int width,
        const int height,
        const float *__restrict__ input,
        float *__restrict__ output)
    {
        const int idy = blockIdx.y * blockDim.y + threadIdx.y;
        const int idx = blockIdx.x * blockDim.x + threadIdx.x;
        if (idy < height && idx < width)
        {
            int cnt[16] = {0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0};
            for (int offsety = -2; offsety <= 2; ++offsety)
            {
                const int py = idy + offsety;
                if (0 <= py && py < height)
                    for (int offsetx = -2; offsetx <= 2; ++offsetx)
                    {
                        const int px = idx + offsetx;
                        if (0 <= px && px < width)
                            ++cnt[(int)input[py * width + px]];
                    }
            }
            const int n = (min(idx, 2) + 1 + min(width - idx, 2)) * (min(idy, 2) + 1 + min(height - idy, 2));
            double ans = mylog[n], n_inv = 1.0 / n;
            for (int i = 0; i < 16; ++i)
                ans -= mylog[cnt[i]] * cnt[i] * n_inv;
            output[idy * width + idx] = ans;
        }
    }

    static void cudaCallback(
        int width,
        int height,
        float *sample,
        float **result)
    {
        float *input_d, *output_d;

        CHECK(hipMalloc((void **)&input_d, sizeof(float) * width * height));
        CHECK(hipMalloc((void **)&output_d, sizeof(float) * width * height));
        CHECK(hipMemcpy(input_d, sample, sizeof(float) * width * height, hipMemcpyHostToDevice));

        const int
            BLOCK_DIM_X = 32,
            BLOCK_DIM_Y = 32;

        const dim3
            blockDim(BLOCK_DIM_X, BLOCK_DIM_Y),
            gridDim(divup(width, BLOCK_DIM_X), divup(height, BLOCK_DIM_Y));

        cudaCallbackKernel<<<
            gridDim,
            blockDim>>>(
            width,
            height,
            input_d,
            output_d);

        *result = (float *)malloc(sizeof(float) * width * height);
        CHECK(hipMemcpy(*result, output_d, sizeof(float) * width * height, hipMemcpyDeviceToHost));
        CHECK(hipFree(input_d));
        CHECK(hipFree(output_d));
    }
} // namespace v4
namespace v5 //cuda 预处理log到texure memory
{
    static texture<float> mylog_tex;
    static __device__ float mylog[26];
    static struct InitPlogp
    {
        InitPlogp()
        {
            float mylog_h[26] = {
                0.0,
                log(1.0),
                log(2.0),
                log(3.0),
                log(4.0),
                log(5.0),
                log(6.0),
                log(7.0),
                log(8.0),
                log(9.0),
                log(10.0),
                log(11.0),
                log(12.0),
                log(13.0),
                log(14.0),
                log(15.0),
                log(16.0),
                log(17.0),
                log(18.0),
                log(19.0),
                log(20.0),
                log(21.0),
                log(22.0),
                log(23.0),
                log(24.0),
                log(25.0)},
                  *mylog_d;
            hipMemcpyToSymbol(HIP_SYMBOL(mylog), mylog_h, sizeof(float) * 26);
            hipGetSymbolAddress((void **)&mylog_d, mylog);
            hipBindTexture(0, mylog_tex, mylog_d);
        }
    } tmpInit;

    static __global__ void cudaCallbackKernel(
        const int width,
        const int height,
        const float *__restrict__ input,
        float *__restrict__ output)
    {
        const int idy = blockIdx.y * blockDim.y + threadIdx.y;
        const int idx = blockIdx.x * blockDim.x + threadIdx.x;
        if (idy < height && idx < width)
        {
            int cnt[16] = {0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0};
            for (int offsety = -2; offsety <= 2; ++offsety)
            {
                const int py = idy + offsety;
                if (0 <= py && py < height)
                    for (int offsetx = -2; offsetx <= 2; ++offsetx)
                    {
                        const int px = idx + offsetx;
                        if (0 <= px && px < width)
                            ++cnt[(int)input[py * width + px]];
                    }
            }
            const int n = (min(idx, 2) + 1 + min(width - idx, 2)) * (min(idy, 2) + 1 + min(height - idy, 2));
            double ans = tex1Dfetch(mylog_tex, n), n_inv = 1.0 / n;
            for (int i = 0; i < 16; ++i)
                ans -= tex1Dfetch(mylog_tex, cnt[i]) * cnt[i] * n_inv;
            output[idy * width + idx] = ans;
        }
    }

    static void cudaCallback(
        int width,
        int height,
        float *sample,
        float **result)
    {
        float *input_d, *output_d;

        CHECK(hipMalloc((void **)&input_d, sizeof(float) * width * height));
        CHECK(hipMalloc((void **)&output_d, sizeof(float) * width * height));
        CHECK(hipMemcpy(input_d, sample, sizeof(float) * width * height, hipMemcpyHostToDevice));

        const int
            BLOCK_DIM_X = 32,
            BLOCK_DIM_Y = 32;

        const dim3
            blockDim(BLOCK_DIM_X, BLOCK_DIM_Y),
            gridDim(divup(width, BLOCK_DIM_X), divup(height, BLOCK_DIM_Y));

        cudaCallbackKernel<<<
            gridDim,
            blockDim>>>(
            width,
            height,
            input_d,
            output_d);

        *result = (float *)malloc(sizeof(float) * width * height);
        CHECK(hipMemcpy(*result, output_d, sizeof(float) * width * height, hipMemcpyDeviceToHost));
        CHECK(hipFree(input_d));
        CHECK(hipFree(output_d));
    }
} // namespace v5
namespace v6 //cuda 预处理log到寄存器+使用更小的整型类型
{
    static __global__ void cudaCallbackKernel(
        const int width,
        const int height,
        const float *__restrict__ input,
        float *__restrict__ output)
    {
        const int idy = blockIdx.y * blockDim.y + threadIdx.y;
        const int idx = blockIdx.x * blockDim.x + threadIdx.x;
        if (idy < height && idx < width)
        {
            signed char cnt[16] = {0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0};
            for (signed char offsety = -2; offsety <= 2; ++offsety)
            {
                const int py = idy + offsety;
                if (0 <= py && py < height)
                    for (signed char offsetx = -2; offsetx <= 2; ++offsetx)
                    {
                        const int px = idx + offsetx;
                        if (0 <= px && px < width)
                            ++cnt[(signed char)input[py * width + px]];
                    }
            }
            const double mylog[26] = {
                0.0,
                log(1.0),
                log(2.0),
                log(3.0),
                log(4.0),
                log(5.0),
                log(6.0),
                log(7.0),
                log(8.0),
                log(9.0),
                log(10.0),
                log(11.0),
                log(12.0),
                log(13.0),
                log(14.0),
                log(15.0),
                log(16.0),
                log(17.0),
                log(18.0),
                log(19.0),
                log(20.0),
                log(21.0),
                log(22.0),
                log(23.0),
                log(24.0),
                log(25.0)};

            const signed char n = (min(idx, 2) + 1 + min(width - idx, 2)) * (min(idy, 2) + 1 + min(height - idy, 2));
            double ans = mylog[n], n_inv = 1.0 / n;
            for (signed char i = 0; i < 16; ++i)
                ans -= mylog[cnt[i]] * cnt[i] * n_inv;
            output[idy * width + idx] = ans;
        }
    }

    static void cudaCallback(
        int width,
        int height,
        float *sample,
        float **result)
    {
        float *input_d, *output_d;

        CHECK(hipMalloc((void **)&input_d, sizeof(float) * width * height));
        CHECK(hipMalloc((void **)&output_d, sizeof(float) * width * height));
        CHECK(hipMemcpy(input_d, sample, sizeof(float) * width * height, hipMemcpyHostToDevice));

        const int
            BLOCK_DIM_X = 32,
            BLOCK_DIM_Y = 32;

        const dim3
            blockDim(BLOCK_DIM_X, BLOCK_DIM_Y),
            gridDim(divup(width, BLOCK_DIM_X), divup(height, BLOCK_DIM_Y));

        cudaCallbackKernel<<<
            gridDim,
            blockDim>>>(
            width,
            height,
            input_d,
            output_d);

        *result = (float *)malloc(sizeof(float) * width * height);
        CHECK(hipMemcpy(*result, output_d, sizeof(float) * width * height, hipMemcpyDeviceToHost));
        CHECK(hipFree(input_d));
        CHECK(hipFree(output_d));
    }
} // namespace v6
namespace v7 //cuda 预处理log到寄存器+使用更小的整型类型+使用更小的浮点类型
{
    static __global__ void cudaCallbackKernel(
        const int width,
        const int height,
        const float *__restrict__ input,
        float *__restrict__ output)
    {
        const int idy = blockIdx.y * blockDim.y + threadIdx.y;
        const int idx = blockIdx.x * blockDim.x + threadIdx.x;
        if (idy < height && idx < width)
        {
            signed char cnt[16] = {0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0};
            for (signed char offsety = -2; offsety <= 2; ++offsety)
            {
                const int py = idy + offsety;
                if (0 <= py && py < height)
                    for (signed char offsetx = -2; offsetx <= 2; ++offsetx)
                    {
                        const int px = idx + offsetx;
                        if (0 <= px && px < width)
                            ++cnt[(signed char)input[py * width + px]];
                    }
            }
            const float mylog[26] = {
                0.0,
                log(1.0),
                log(2.0),
                log(3.0),
                log(4.0),
                log(5.0),
                log(6.0),
                log(7.0),
                log(8.0),
                log(9.0),
                log(10.0),
                log(11.0),
                log(12.0),
                log(13.0),
                log(14.0),
                log(15.0),
                log(16.0),
                log(17.0),
                log(18.0),
                log(19.0),
                log(20.0),
                log(21.0),
                log(22.0),
                log(23.0),
                log(24.0),
                log(25.0)};

            const signed char n = (min(idx, 2) + 1 + min(width - idx, 2)) * (min(idy, 2) + 1 + min(height - idy, 2));
            double ans = mylog[n], n_inv = 1.0 / n;
            for (signed char i = 0; i < 16; ++i)
                ans -= mylog[cnt[i]] * n_inv * cnt[i];
            output[idy * width + idx] = ans;
        }
    }

    static void cudaCallback(
        int width,
        int height,
        float *sample,
        float **result)
    {
        float *input_d, *output_d;

        CHECK(hipMalloc((void **)&input_d, sizeof(float) * width * height));
        CHECK(hipMalloc((void **)&output_d, sizeof(float) * width * height));
        CHECK(hipMemcpy(input_d, sample, sizeof(float) * width * height, hipMemcpyHostToDevice));

        const int
            BLOCK_DIM_X = 32,
            BLOCK_DIM_Y = 32;

        const dim3
            blockDim(BLOCK_DIM_X, BLOCK_DIM_Y),
            gridDim(divup(width, BLOCK_DIM_X), divup(height, BLOCK_DIM_Y));

        cudaCallbackKernel<<<
            gridDim,
            blockDim>>>(
            width,
            height,
            input_d,
            output_d);

        *result = (float *)malloc(sizeof(float) * width * height);
        CHECK(hipMemcpy(*result, output_d, sizeof(float) * width * height, hipMemcpyDeviceToHost));
        CHECK(hipFree(input_d));
        CHECK(hipFree(output_d));
    }
} // namespace v7
namespace v8 //cuda 预处理log到寄存器+使用更小的整型类型+使用更小的浮点类型+使用texure memory优化读入
{
    static texture<float, 2> input_tex;

    static __global__ void cudaCallbackKernel(
        const int width,
        const int height,
        float *__restrict__ output)
    {
        const int idy = blockIdx.y * blockDim.y + threadIdx.y;
        const int idx = blockIdx.x * blockDim.x + threadIdx.x;
        if (idy < height && idx < width)
        {
            signed char cnt[16] = {0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0};
            for (signed char offsety = -2; offsety <= 2; ++offsety)
            {
                const int py = idy + offsety;
                if (0 <= py && py < height)
                    for (signed char offsetx = -2; offsetx <= 2; ++offsetx)
                    {
                        const int px = idx + offsetx;
                        if (0 <= px && px < width)
                            ++cnt[(signed char)tex2D(input_tex, px, py)];
                    }
            }
            const float mylog[26] = {
                0.0,
                log(1.0),
                log(2.0),
                log(3.0),
                log(4.0),
                log(5.0),
                log(6.0),
                log(7.0),
                log(8.0),
                log(9.0),
                log(10.0),
                log(11.0),
                log(12.0),
                log(13.0),
                log(14.0),
                log(15.0),
                log(16.0),
                log(17.0),
                log(18.0),
                log(19.0),
                log(20.0),
                log(21.0),
                log(22.0),
                log(23.0),
                log(24.0),
                log(25.0)};

            const signed char n = (min(idx, 2) + 1 + min(width - idx, 2)) * (min(idy, 2) + 1 + min(height - idy, 2));
            double ans = mylog[n], n_inv = 1.0 / n;
            for (signed char i = 0; i < 16; ++i)
                ans -= mylog[cnt[i]] * n_inv * cnt[i];
            output[idy * width + idx] = ans;
        }
    }

    static void cudaCallback(
        int width,
        int height,
        float *sample,
        float **result)
    {
        float *output_d;

        CHECK(hipMalloc((void **)&output_d, sizeof(float) * width * height));
        hipChannelFormatDesc channelDesc = hipCreateChannelDesc<float>();
        hipArray *cuArray;
        CHECK(hipMallocArray(&cuArray, &channelDesc, width, height));
        CHECK(hipMemcpyToArray(cuArray, 0, 0, sample, sizeof(float) * width * height, hipMemcpyHostToDevice));
        CHECK(hipBindTextureToArray(input_tex, cuArray));

        const int
            BLOCK_DIM_X = 32,
            BLOCK_DIM_Y = 32;

        const dim3
            blockDim(BLOCK_DIM_X, BLOCK_DIM_Y),
            gridDim(divup(width, BLOCK_DIM_X), divup(height, BLOCK_DIM_Y));

        cudaCallbackKernel<<<
            gridDim,
            blockDim>>>(
            width,
            height,
            output_d);

        *result = (float *)malloc(sizeof(float) * width * height);
        CHECK(hipMemcpy(*result, output_d, sizeof(float) * width * height, hipMemcpyDeviceToHost));
        CHECK(hipFreeArray(cuArray));
        CHECK(hipFree(output_d));
    }
} // namespace v8
namespace v9 //cuda 预处理log到寄存器+使用更小的整型类型+使用更小的浮点类型+使用shared memory优化读入
{
    template <
        int BLOCK_DIM_X,
        int BLOCK_DIM_Y>
    static __global__ void cudaCallbackKernel(
        const int width,
        const int height,
        const float *__restrict__ input,
        float *__restrict__ output)
    {
        const int idy = blockIdx.y * (BLOCK_DIM_Y - 4) + threadIdx.y - 2;
        const int idx = blockIdx.x * (BLOCK_DIM_X - 4) + threadIdx.x - 2;
        __shared__ int input_s[BLOCK_DIM_Y][BLOCK_DIM_X | 1];

        input_s[threadIdx.y][threadIdx.x] = 0 <= idy && idy < height && 0 <= idx && idx < width ? input[idy * width + idx] : 16;

        __syncthreads();

        if (1 < threadIdx.y && threadIdx.y < BLOCK_DIM_Y - 2 &&
            1 < threadIdx.x && threadIdx.x < BLOCK_DIM_X - 2 &&
            idy < height && idx < width)
        {
            signed char cnt[17] = {0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0};
            for (signed char offsety = -2; offsety <= 2; ++offsety)
                for (signed char offsetx = -2; offsetx <= 2; ++offsetx)
                    ++cnt[input_s[threadIdx.y + offsety][threadIdx.x + offsetx]];

            const float mylog[26] = {
                0.0,
                log(1.0),
                log(2.0),
                log(3.0),
                log(4.0),
                log(5.0),
                log(6.0),
                log(7.0),
                log(8.0),
                log(9.0),
                log(10.0),
                log(11.0),
                log(12.0),
                log(13.0),
                log(14.0),
                log(15.0),
                log(16.0),
                log(17.0),
                log(18.0),
                log(19.0),
                log(20.0),
                log(21.0),
                log(22.0),
                log(23.0),
                log(24.0),
                log(25.0)};

            const signed char n = (min(idx, 2) + 1 + min(width - idx, 2)) * (min(idy, 2) + 1 + min(height - idy, 2));
            double ans = mylog[n], n_inv = 1.0 / n;
            for (signed char i = 0; i < 16; ++i)
                ans -= mylog[cnt[i]] * n_inv * cnt[i];
            output[idy * width + idx] = ans;
        }
    }

    static void cudaCallback(
        int width,
        int height,
        float *sample,
        float **result)
    {
        float *input_d, *output_d;

        CHECK(hipMalloc((void **)&input_d, sizeof(float) * width * height));
        CHECK(hipMalloc((void **)&output_d, sizeof(float) * width * height));
        CHECK(hipMemcpy(input_d, sample, sizeof(float) * width * height, hipMemcpyHostToDevice));

        const int
            BLOCK_DIM_X = 32,
            BLOCK_DIM_Y = 32;

        const dim3
            blockDim(BLOCK_DIM_X, BLOCK_DIM_Y),
            gridDim(divup(width, BLOCK_DIM_X - 4), divup(height, BLOCK_DIM_Y - 4));

        cudaCallbackKernel<
            BLOCK_DIM_X,
            BLOCK_DIM_Y><<<
            gridDim,
            blockDim>>>(
            width,
            height,
            input_d,
            output_d);

        *result = (float *)malloc(sizeof(float) * width * height);
        CHECK(hipMemcpy(*result, output_d, sizeof(float) * width * height, hipMemcpyDeviceToHost));
        CHECK(hipFree(input_d));
        CHECK(hipFree(output_d));
    }
} // namespace v9
namespace v10 //openmp baseline
{
    static void cudaCallback(
        int width,
        int height,
        float *sample,
        float **result)
    {
        *result = (float *)malloc(sizeof(float) * width * height);
#pragma omp parallel for
        for (int pos = 0; pos < width * height; ++pos)
        {
            const int
                idy = pos / width,
                idx = pos - idy * width;
            int cnt[16] = {0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0};
            for (int offsety = -2; offsety <= 2; ++offsety)
                for (int offsetx = -2; offsetx <= 2; ++offsetx)
                {
                    const int py = idy + offsety, px = idx + offsetx;
                    if (0 <= py && py < height && 0 <= px && px < width)
                        ++cnt[(int)sample[py * width + px]];
                }
            double n = (min(idx, 2) + 1 + min(width - idx, 2)) * (min(idy, 2) + 1 + min(height - idy, 2)), ans = log(n), n_inv = 1.0 / n;
            for (int i = 0; i < 16; ++i)
                if (cnt[i])
                    ans -= log((double)cnt[i]) * cnt[i] * n_inv;
            (*result)[pos] = ans;
        }
    }
} // namespace v10
namespace v11 //openmp 预处理log到寄存器
{
    static void cudaCallback(
        int width,
        int height,
        float *sample,
        float **result)
    {
        *result = (float *)malloc(sizeof(float) * width * height);
#pragma omp parallel for
        for (int pos = 0; pos < width * height; ++pos)
        {
            const int
                idy = pos / width,
                idx = pos - idy * width;
            int cnt[16] = {0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0};
            for (int offsety = -2; offsety <= 2; ++offsety)
                for (int offsetx = -2; offsetx <= 2; ++offsetx)
                {
                    const int py = idy + offsety, px = idx + offsetx;
                    if (0 <= py && py < height && 0 <= px && px < width)
                        ++cnt[(int)sample[py * width + px]];
                }
            const double mylog[26] = {
                0.0,
                log(1.0),
                log(2.0),
                log(3.0),
                log(4.0),
                log(5.0),
                log(6.0),
                log(7.0),
                log(8.0),
                log(9.0),
                log(10.0),
                log(11.0),
                log(12.0),
                log(13.0),
                log(14.0),
                log(15.0),
                log(16.0),
                log(17.0),
                log(18.0),
                log(19.0),
                log(20.0),
                log(21.0),
                log(22.0),
                log(23.0),
                log(24.0),
                log(25.0)};
            const int n = (min(idx, 2) + 1 + min(width - idx, 2)) * (min(idy, 2) + 1 + min(height - idy, 2));
            double ans = mylog[n], n_inv = 1.0 / n;
            for (int i = 0; i < 16; ++i)
                ans -= mylog[cnt[i]] * n_inv * cnt[i];
            (*result)[pos] = ans;
        }
    }
} // namespace v11
namespace v12 //openmp 预处理log到寄存器+使用更小的类型
{
    static void cudaCallback(
        int width,
        int height,
        float *sample,
        float **result)
    {
        *result = (float *)malloc(sizeof(float) * width * height);
#pragma omp parallel for
        for (int pos = 0; pos < width * height; ++pos)
        {
            const int
                idy = pos / width,
                idx = pos - idy * width;
            signed char cnt[16] = {0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0};
            for (signed char offsety = -2; offsety <= 2; ++offsety)
                for (signed char offsetx = -2; offsetx <= 2; ++offsetx)
                {
                    const int py = idy + offsety, px = idx + offsetx;
                    if (0 <= py && py < height && 0 <= px && px < width)
                        ++cnt[(signed char)sample[py * width + px]];
                }
            const float mylog[26] = {
                0.0,
                log(1.0),
                log(2.0),
                log(3.0),
                log(4.0),
                log(5.0),
                log(6.0),
                log(7.0),
                log(8.0),
                log(9.0),
                log(10.0),
                log(11.0),
                log(12.0),
                log(13.0),
                log(14.0),
                log(15.0),
                log(16.0),
                log(17.0),
                log(18.0),
                log(19.0),
                log(20.0),
                log(21.0),
                log(22.0),
                log(23.0),
                log(24.0),
                log(25.0)};
            const signed char n = (min(idx, 2) + 1 + min(width - idx, 2)) * (min(idy, 2) + 1 + min(height - idy, 2));
            double ans = mylog[n], n_inv = 1.0 / n;
            for (signed char i = 0; i < 16; ++i)
                ans -= mylog[cnt[i]] * n_inv * cnt[i];
            (*result)[pos] = ans;
        }
    }
} // namespace v12
namespace v13 //openmp 预处理log到寄存器+使用更小的类型+预处理前缀和
{
    static void cudaCallback(
        int width,
        int height,
        float *sample,
        float **result)
    {
        *result = (float *)malloc(sizeof(float) * width * height);
        int *sum[16];
#pragma omp parallel for
        for (int i = 0; i < 16; ++i)
        {
            int *p = (int *)malloc(sizeof(int) * (width + 5) * (height + 5));
            for (int pos = 0; pos < (width + 5) * (height + 5); ++pos)
            {
                const int idy = pos / (width + 5), idx = pos - idy * (width + 5);
                if (idy && idx)
                {
                    p[pos] = p[(idy - 1) * (width + 5) + idx] + p[idy * (width + 5) + idx - 1] - p[(idy - 1) * (width + 5) + (idx - 1)];
                    const int py = idy - 3, px = idx - 3;
                    if (0 <= py && py < height && 0 <= px && px < width && i == sample[py * width + px])
                        ++p[pos];
                }
                else
                    p[pos] = 0;
            }
            sum[i] = p;
        }
#pragma omp parallel for
        for (int pos = 0; pos < width * height; ++pos)
        {
            const int
                idy = pos / width,
                idx = pos - idy * width;
            const float mylog[26] = {
                0.0,
                log(1.0),
                log(2.0),
                log(3.0),
                log(4.0),
                log(5.0),
                log(6.0),
                log(7.0),
                log(8.0),
                log(9.0),
                log(10.0),
                log(11.0),
                log(12.0),
                log(13.0),
                log(14.0),
                log(15.0),
                log(16.0),
                log(17.0),
                log(18.0),
                log(19.0),
                log(20.0),
                log(21.0),
                log(22.0),
                log(23.0),
                log(24.0),
                log(25.0)};
            const signed char n = (min(idx, 2) + 1 + min(width - idx, 2)) * (min(idy, 2) + 1 + min(height - idy, 2));
            double ans = mylog[n], n_inv = 1.0 / n;
            for (signed char i = 0; i < 16; ++i)
            {
                const signed char cnti = sum[i][(idy + 5) * (width + 5) + idx + 5] - sum[i][(idy + 5) * (width + 5) + idx] - sum[i][idy * (width + 5) + idx + 5] + sum[i][idy * (width + 5) + idx];
                ans -= mylog[cnti] * n_inv * cnti;
            }
            (*result)[pos] = ans;
        }
        for (int i = 0; i < 16; ++i)
            free(sum[i]);
    }
} // namespace v13
namespace v14 //cuda+openmp 多卡，基于v7、v9、v11
{
    static void cudaCallback(
        int width,
        int height,
        float *sample,
        float **result)
    {
        int num_threads = 0;
        CHECK(hipGetDeviceCount(&num_threads));
        if (num_threads > height - 4)
            num_threads = height - 4;
        if (num_threads < 1)
            return v11::cudaCallback(width, height, sample, result);
        if (num_threads < 2 || width * height < 1e5)
            return v7::cudaCallback(width, height, sample, result);
        *result = (float *)malloc(sizeof(float) * width * height);
#pragma omp parallel num_threads(num_threads)
        {
            int thread_num = omp_get_thread_num(),
                thread_hgt = (height - 4) / num_threads,
                thread_beg = thread_hgt * thread_num + 2;
            if (thread_num == num_threads - 1)
                thread_hgt = height - 2 - thread_beg;
            float *thread_result;
            CHECK(hipSetDevice(thread_num));
            v9::cudaCallback(
                width,
                thread_hgt + 4,
                sample + width * (thread_beg - 2),
                &thread_result);
            float
                *dst = (*result) + width * thread_beg,
                *src = thread_result + width * 2;
            if (thread_num == 0)
                dst -= width * 2, src -= width * 2, thread_hgt += 2;
            if (thread_num == num_threads - 1)
                thread_hgt += 2;
            memcpy(
                dst,
                src,
                sizeof(float) * width * thread_hgt);
            free(thread_result);
        }
    }
} // namespace v14
void cudaCallback(
    int width,
    int height,
    float *sample,
    float **result)
{
    v14::cudaCallback(width, height, sample, result);
}
