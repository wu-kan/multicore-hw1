#include "hip/hip_runtime.h"
#include "core.h"

__forceinline__ __device__ __host__ int
neighborhood_size(int x, int y, int width, int height)
{
    switch (min(min(x, width - x), 2) << 2 | min(min(y, height - y), 2))
    {
    case 0:
        return 9;
    case 1:
    case 4:
        return 12;
    case 2:
    case 8:
        return 15;
    case 5:
        return 16;
    case 6:
    case 9:
        return 20;
    default:
        return 25;
    }
}

namespace v0 //cuda baseline
{
    __global__ void cudaCallbackKernel(
        const int width,
        const int height,
        const float *__restrict__ input,
        float *__restrict__ output)
    {
        const int idy = blockIdx.y * blockDim.y + threadIdx.y;
        const int idx = blockIdx.x * blockDim.x + threadIdx.x;
        if (idy < height && idx < width)
        {
            int cnt[16] = {0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0};
            for (int offsety = -2; offsety <= 2; ++offsety)
            {
                const int py = idy + offsety;
                if (0 <= py && py < height)
                    for (int offsetx = -2; offsetx <= 2; ++offsetx)
                    {
                        const int px = idx + offsetx;
                        if (0 <= px && px < width)
                            ++cnt[(int)input[py * width + px]];
                    }
            }
            double n = neighborhood_size(idx, idy, width, height), ans = log(n);
            for (int i = 0; i < 16; ++i)
                if (cnt[i])
                    ans -= log((double)cnt[i]) * cnt[i] / n;
            output[idy * width + idx] = ans;
        }
    }

    void cudaCallback(
        int width,
        int height,
        float *sample,
        float **result)
    {
        float *input_d, *output_d;

        CHECK(hipMalloc((void **)&input_d, sizeof(float) * width * height));
        CHECK(hipMalloc((void **)&output_d, sizeof(float) * width * height));
        CHECK(hipMemcpy(input_d, sample, sizeof(float) * width * height, hipMemcpyHostToDevice));

        const int
            BLOCK_DIM_X = 32,
            BLOCK_DIM_Y = 32;

        const dim3
            blockDim(BLOCK_DIM_X, BLOCK_DIM_Y),
            gridDim(divup(width, BLOCK_DIM_X), divup(height, BLOCK_DIM_Y));

        cudaCallbackKernel<<<
            gridDim,
            blockDim>>>(
            width,
            height,
            input_d,
            output_d);

        *result = (float *)malloc(sizeof(float) * width * height);
        CHECK(hipMemcpy(*result, output_d, sizeof(float) * width * height, hipMemcpyDeviceToHost));
        CHECK(hipFree(input_d));
        CHECK(hipFree(output_d));
    }
} // namespace v0
namespace v1 //cuda 预处理log到寄存器
{
    __global__ void cudaCallbackKernel(
        const int width,
        const int height,
        const float *__restrict__ input,
        float *__restrict__ output)
    {
        const int idy = blockIdx.y * blockDim.y + threadIdx.y;
        const int idx = blockIdx.x * blockDim.x + threadIdx.x;
        if (idy < height && idx < width)
        {
            int cnt[16] = {0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0};
            for (int offsety = -2; offsety <= 2; ++offsety)
            {
                const int py = idy + offsety;
                if (0 <= py && py < height)
                    for (int offsetx = -2; offsetx <= 2; ++offsetx)
                    {
                        const int px = idx + offsetx;
                        if (0 <= px && px < width)
                            ++cnt[(int)input[py * width + px]];
                    }
            }
            const double mylog[26] = {
                0.0,
                log(1.0),
                log(2.0),
                log(3.0),
                log(4.0),
                log(5.0),
                log(6.0),
                log(7.0),
                log(8.0),
                log(9.0),
                log(10.0),
                log(11.0),
                log(12.0),
                log(13.0),
                log(14.0),
                log(15.0),
                log(16.0),
                log(17.0),
                log(18.0),
                log(19.0),
                log(20.0),
                log(21.0),
                log(22.0),
                log(23.0),
                log(24.0),
                log(25.0)};

            const int n = neighborhood_size(idx, idy, width, height);
            double ans = mylog[n];
            for (int i = 0; i < 16; ++i)
                ans -= mylog[cnt[i]] * cnt[i] / n;
            output[idy * width + idx] = ans;
        }
    }

    void cudaCallback(
        int width,
        int height,
        float *sample,
        float **result)
    {
        float *input_d, *output_d;

        CHECK(hipMalloc((void **)&input_d, sizeof(float) * width * height));
        CHECK(hipMalloc((void **)&output_d, sizeof(float) * width * height));
        CHECK(hipMemcpy(input_d, sample, sizeof(float) * width * height, hipMemcpyHostToDevice));

        const int
            BLOCK_DIM_X = 32,
            BLOCK_DIM_Y = 32;

        const dim3
            blockDim(BLOCK_DIM_X, BLOCK_DIM_Y),
            gridDim(divup(width, BLOCK_DIM_X), divup(height, BLOCK_DIM_Y));

        cudaCallbackKernel<<<
            gridDim,
            blockDim>>>(
            width,
            height,
            input_d,
            output_d);

        *result = (float *)malloc(sizeof(float) * width * height);
        CHECK(hipMemcpy(*result, output_d, sizeof(float) * width * height, hipMemcpyDeviceToHost));
        CHECK(hipFree(input_d));
        CHECK(hipFree(output_d));
    }
} // namespace v1
namespace v2 //cuda 预处理log到shared memory
{
    __global__ void cudaCallbackKernel(
        const int width,
        const int height,
        const float *__restrict__ input,
        float *__restrict__ output)
    {
        const int idy = blockIdx.y * blockDim.y + threadIdx.y;
        const int idx = blockIdx.x * blockDim.x + threadIdx.x;

        __shared__ double mylog[26];
        if (threadIdx.y == 0 && threadIdx.x < 26)
            mylog[threadIdx.x] = threadIdx.x == 0 ? 0.0 : log((double)threadIdx.x);
        __syncthreads();
        if (idy < height && idx < width)
        {
            int cnt[16] = {0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0};
            for (int offsety = -2; offsety <= 2; ++offsety)
            {
                const int py = idy + offsety;
                if (0 <= py && py < height)
                    for (int offsetx = -2; offsetx <= 2; ++offsetx)
                    {
                        const int px = idx + offsetx;
                        if (0 <= px && px < width)
                            ++cnt[(int)input[py * width + px]];
                    }
            }
            const int n = neighborhood_size(idx, idy, width, height);
            double ans = mylog[n];
            for (int i = 0; i < 16; ++i)
                ans -= mylog[cnt[i]] * cnt[i] / n;
            output[idy * width + idx] = ans;
        }
    }

    void cudaCallback(
        int width,
        int height,
        float *sample,
        float **result)
    {
        float *input_d, *output_d;

        CHECK(hipMalloc((void **)&input_d, sizeof(float) * width * height));
        CHECK(hipMalloc((void **)&output_d, sizeof(float) * width * height));
        CHECK(hipMemcpy(input_d, sample, sizeof(float) * width * height, hipMemcpyHostToDevice));

        const int
            BLOCK_DIM_X = 32,
            BLOCK_DIM_Y = 32;

        const dim3
            blockDim(BLOCK_DIM_X, BLOCK_DIM_Y),
            gridDim(divup(width, BLOCK_DIM_X), divup(height, BLOCK_DIM_Y));

        cudaCallbackKernel<<<
            gridDim,
            blockDim>>>(
            width,
            height,
            input_d,
            output_d);

        *result = (float *)malloc(sizeof(float) * width * height);
        CHECK(hipMemcpy(*result, output_d, sizeof(float) * width * height, hipMemcpyDeviceToHost));
        CHECK(hipFree(input_d));
        CHECK(hipFree(output_d));
    }
} // namespace v2
namespace v3 //cuda 预处理log到constant memory
{
    __constant__ double mylog[26];
    struct InitPlogp
    {
        InitPlogp()
        {
            const double mylog_h[26] = {
                0.0,
                log(1.0),
                log(2.0),
                log(3.0),
                log(4.0),
                log(5.0),
                log(6.0),
                log(7.0),
                log(8.0),
                log(9.0),
                log(10.0),
                log(11.0),
                log(12.0),
                log(13.0),
                log(14.0),
                log(15.0),
                log(16.0),
                log(17.0),
                log(18.0),
                log(19.0),
                log(20.0),
                log(21.0),
                log(22.0),
                log(23.0),
                log(24.0),
                log(25.0)};
            hipMemcpyToSymbol(HIP_SYMBOL(mylog), mylog_h, sizeof(double) * 26);
        }
    } tmpInit;

    __global__ void cudaCallbackKernel(
        const int width,
        const int height,
        const float *__restrict__ input,
        float *__restrict__ output)
    {
        const int idy = blockIdx.y * blockDim.y + threadIdx.y;
        const int idx = blockIdx.x * blockDim.x + threadIdx.x;
        if (idy < height && idx < width)
        {
            int cnt[16] = {0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0};
            for (int offsety = -2; offsety <= 2; ++offsety)
            {
                const int py = idy + offsety;
                if (0 <= py && py < height)
                    for (int offsetx = -2; offsetx <= 2; ++offsetx)
                    {
                        const int px = idx + offsetx;
                        if (0 <= px && px < width)
                            ++cnt[(int)input[py * width + px]];
                    }
            }
            const int n = neighborhood_size(idx, idy, width, height);
            double ans = mylog[n];
            for (int i = 0; i < 16; ++i)
                ans -= mylog[cnt[i]] * cnt[i] / n;
            output[idy * width + idx] = ans;
        }
    }

    void cudaCallback(
        int width,
        int height,
        float *sample,
        float **result)
    {
        float *input_d, *output_d;

        CHECK(hipMalloc((void **)&input_d, sizeof(float) * width * height));
        CHECK(hipMalloc((void **)&output_d, sizeof(float) * width * height));
        CHECK(hipMemcpy(input_d, sample, sizeof(float) * width * height, hipMemcpyHostToDevice));

        const int
            BLOCK_DIM_X = 32,
            BLOCK_DIM_Y = 32;

        const dim3
            blockDim(BLOCK_DIM_X, BLOCK_DIM_Y),
            gridDim(divup(width, BLOCK_DIM_X), divup(height, BLOCK_DIM_Y));

        cudaCallbackKernel<<<
            gridDim,
            blockDim>>>(
            width,
            height,
            input_d,
            output_d);

        *result = (float *)malloc(sizeof(float) * width * height);
        CHECK(hipMemcpy(*result, output_d, sizeof(float) * width * height, hipMemcpyDeviceToHost));
        CHECK(hipFree(input_d));
        CHECK(hipFree(output_d));
    }
} // namespace v3
namespace v4 //cuda 预处理log到device memory
{
    __device__ double mylog[26];
    struct InitPlogp
    {
        InitPlogp()
        {
            const double mylog_h[26] = {
                0.0,
                log(1.0),
                log(2.0),
                log(3.0),
                log(4.0),
                log(5.0),
                log(6.0),
                log(7.0),
                log(8.0),
                log(9.0),
                log(10.0),
                log(11.0),
                log(12.0),
                log(13.0),
                log(14.0),
                log(15.0),
                log(16.0),
                log(17.0),
                log(18.0),
                log(19.0),
                log(20.0),
                log(21.0),
                log(22.0),
                log(23.0),
                log(24.0),
                log(25.0)};
            hipMemcpyToSymbol(HIP_SYMBOL(mylog), mylog_h, sizeof(double) * 26);
        }
    } tmpInit;

    __global__ void cudaCallbackKernel(
        const int width,
        const int height,
        const float *__restrict__ input,
        float *__restrict__ output)
    {
        const int idy = blockIdx.y * blockDim.y + threadIdx.y;
        const int idx = blockIdx.x * blockDim.x + threadIdx.x;
        if (idy < height && idx < width)
        {
            int cnt[16] = {0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0};
            for (int offsety = -2; offsety <= 2; ++offsety)
            {
                const int py = idy + offsety;
                if (0 <= py && py < height)
                    for (int offsetx = -2; offsetx <= 2; ++offsetx)
                    {
                        const int px = idx + offsetx;
                        if (0 <= px && px < width)
                            ++cnt[(int)input[py * width + px]];
                    }
            }
            const int n = neighborhood_size(idx, idy, width, height);
            double ans = mylog[n];
            for (int i = 0; i < 16; ++i)
                ans -= mylog[cnt[i]] * cnt[i] / n;
            output[idy * width + idx] = ans;
        }
    }

    void cudaCallback(
        int width,
        int height,
        float *sample,
        float **result)
    {
        float *input_d, *output_d;

        CHECK(hipMalloc((void **)&input_d, sizeof(float) * width * height));
        CHECK(hipMalloc((void **)&output_d, sizeof(float) * width * height));
        CHECK(hipMemcpy(input_d, sample, sizeof(float) * width * height, hipMemcpyHostToDevice));

        const int
            BLOCK_DIM_X = 32,
            BLOCK_DIM_Y = 32;

        const dim3
            blockDim(BLOCK_DIM_X, BLOCK_DIM_Y),
            gridDim(divup(width, BLOCK_DIM_X), divup(height, BLOCK_DIM_Y));

        cudaCallbackKernel<<<
            gridDim,
            blockDim>>>(
            width,
            height,
            input_d,
            output_d);

        *result = (float *)malloc(sizeof(float) * width * height);
        CHECK(hipMemcpy(*result, output_d, sizeof(float) * width * height, hipMemcpyDeviceToHost));
        CHECK(hipFree(input_d));
        CHECK(hipFree(output_d));
    }
} // namespace v4
namespace v5 //cuda 预处理log到texure memory
{
    texture<float> mylog_tex;
    __device__ float mylog[26];
    struct InitPlogp
    {
        InitPlogp()
        {
            float mylog_h[26] = {
                0.0,
                log(1.0),
                log(2.0),
                log(3.0),
                log(4.0),
                log(5.0),
                log(6.0),
                log(7.0),
                log(8.0),
                log(9.0),
                log(10.0),
                log(11.0),
                log(12.0),
                log(13.0),
                log(14.0),
                log(15.0),
                log(16.0),
                log(17.0),
                log(18.0),
                log(19.0),
                log(20.0),
                log(21.0),
                log(22.0),
                log(23.0),
                log(24.0),
                log(25.0)},
                  *mylog_d;
            hipMemcpyToSymbol(HIP_SYMBOL(mylog), mylog_h, sizeof(float) * 26);
            hipGetSymbolAddress((void **)&mylog_d, mylog);
            hipBindTexture(0, mylog_tex, mylog_d);
        }
    } tmpInit;

    __global__ void cudaCallbackKernel(
        const int width,
        const int height,
        const float *__restrict__ input,
        float *__restrict__ output)
    {
        const int idy = blockIdx.y * blockDim.y + threadIdx.y;
        const int idx = blockIdx.x * blockDim.x + threadIdx.x;
        if (idy < height && idx < width)
        {
            int cnt[16] = {0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0};
            for (int offsety = -2; offsety <= 2; ++offsety)
            {
                const int py = idy + offsety;
                if (0 <= py && py < height)
                    for (int offsetx = -2; offsetx <= 2; ++offsetx)
                    {
                        const int px = idx + offsetx;
                        if (0 <= px && px < width)
                            ++cnt[(int)input[py * width + px]];
                    }
            }
            const int n = neighborhood_size(idx, idy, width, height);
            double ans = tex1Dfetch(mylog_tex, n);
            for (int i = 0; i < 16; ++i)
                ans -= tex1Dfetch(mylog_tex, cnt[i]) * cnt[i] / n;
            output[idy * width + idx] = ans;
        }
    }

    void cudaCallback(
        int width,
        int height,
        float *sample,
        float **result)
    {
        float *input_d, *output_d;

        CHECK(hipMalloc((void **)&input_d, sizeof(float) * width * height));
        CHECK(hipMalloc((void **)&output_d, sizeof(float) * width * height));
        CHECK(hipMemcpy(input_d, sample, sizeof(float) * width * height, hipMemcpyHostToDevice));

        const int
            BLOCK_DIM_X = 32,
            BLOCK_DIM_Y = 32;

        const dim3
            blockDim(BLOCK_DIM_X, BLOCK_DIM_Y),
            gridDim(divup(width, BLOCK_DIM_X), divup(height, BLOCK_DIM_Y));

        cudaCallbackKernel<<<
            gridDim,
            blockDim>>>(
            width,
            height,
            input_d,
            output_d);

        *result = (float *)malloc(sizeof(float) * width * height);
        CHECK(hipMemcpy(*result, output_d, sizeof(float) * width * height, hipMemcpyDeviceToHost));
        CHECK(hipFree(input_d));
        CHECK(hipFree(output_d));
    }
} // namespace v5
namespace v6 //cuda 预处理log到寄存器+使用更小的整型类型
{
    __global__ void cudaCallbackKernel(
        const int width,
        const int height,
        const float *__restrict__ input,
        float *__restrict__ output)
    {
        const int idy = blockIdx.y * blockDim.y + threadIdx.y;
        const int idx = blockIdx.x * blockDim.x + threadIdx.x;
        if (idy < height && idx < width)
        {
            signed char cnt[16] = {0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0};
            for (signed char offsety = -2; offsety <= 2; ++offsety)
            {
                const int py = idy + offsety;
                if (0 <= py && py < height)
                    for (signed char offsetx = -2; offsetx <= 2; ++offsetx)
                    {
                        const int px = idx + offsetx;
                        if (0 <= px && px < width)
                            ++cnt[(signed char)input[py * width + px]];
                    }
            }
            const double mylog[26] = {
                0.0,
                log(1.0),
                log(2.0),
                log(3.0),
                log(4.0),
                log(5.0),
                log(6.0),
                log(7.0),
                log(8.0),
                log(9.0),
                log(10.0),
                log(11.0),
                log(12.0),
                log(13.0),
                log(14.0),
                log(15.0),
                log(16.0),
                log(17.0),
                log(18.0),
                log(19.0),
                log(20.0),
                log(21.0),
                log(22.0),
                log(23.0),
                log(24.0),
                log(25.0)};

            const signed char n = neighborhood_size(idx, idy, width, height);
            double ans = mylog[n];
            for (signed char i = 0; i < 16; ++i)
                ans -= mylog[cnt[i]] * cnt[i] / n;
            output[idy * width + idx] = ans;
        }
    }

    void cudaCallback(
        int width,
        int height,
        float *sample,
        float **result)
    {
        float *input_d, *output_d;

        CHECK(hipMalloc((void **)&input_d, sizeof(float) * width * height));
        CHECK(hipMalloc((void **)&output_d, sizeof(float) * width * height));
        CHECK(hipMemcpy(input_d, sample, sizeof(float) * width * height, hipMemcpyHostToDevice));

        const int
            BLOCK_DIM_X = 32,
            BLOCK_DIM_Y = 32;

        const dim3
            blockDim(BLOCK_DIM_X, BLOCK_DIM_Y),
            gridDim(divup(width, BLOCK_DIM_X), divup(height, BLOCK_DIM_Y));

        cudaCallbackKernel<<<
            gridDim,
            blockDim>>>(
            width,
            height,
            input_d,
            output_d);

        *result = (float *)malloc(sizeof(float) * width * height);
        CHECK(hipMemcpy(*result, output_d, sizeof(float) * width * height, hipMemcpyDeviceToHost));
        CHECK(hipFree(input_d));
        CHECK(hipFree(output_d));
    }
} // namespace v6
namespace v7 //cuda 预处理log到寄存器+使用更小的整型类型+使用更小的浮点类型
{
    __global__ void cudaCallbackKernel(
        const int width,
        const int height,
        const float *__restrict__ input,
        float *__restrict__ output)
    {
        const int idy = blockIdx.y * blockDim.y + threadIdx.y;
        const int idx = blockIdx.x * blockDim.x + threadIdx.x;
        if (idy < height && idx < width)
        {
            signed char cnt[16] = {0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0};
            for (signed char offsety = -2; offsety <= 2; ++offsety)
            {
                const int py = idy + offsety;
                if (0 <= py && py < height)
                    for (signed char offsetx = -2; offsetx <= 2; ++offsetx)
                    {
                        const int px = idx + offsetx;
                        if (0 <= px && px < width)
                            ++cnt[(signed char)input[py * width + px]];
                    }
            }
            const float mylog[26] = {
                0.0,
                log(1.0),
                log(2.0),
                log(3.0),
                log(4.0),
                log(5.0),
                log(6.0),
                log(7.0),
                log(8.0),
                log(9.0),
                log(10.0),
                log(11.0),
                log(12.0),
                log(13.0),
                log(14.0),
                log(15.0),
                log(16.0),
                log(17.0),
                log(18.0),
                log(19.0),
                log(20.0),
                log(21.0),
                log(22.0),
                log(23.0),
                log(24.0),
                log(25.0)};

            const signed char n = neighborhood_size(idx, idy, width, height);
            double ans = mylog[n], inv_n = 1.0 / n;
            for (signed char i = 0; i < 16; ++i)
                ans -= mylog[cnt[i]] * inv_n * cnt[i];
            output[idy * width + idx] = ans;
        }
    }

    void cudaCallback(
        int width,
        int height,
        float *sample,
        float **result)
    {
        float *input_d, *output_d;

        CHECK(hipMalloc((void **)&input_d, sizeof(float) * width * height));
        CHECK(hipMalloc((void **)&output_d, sizeof(float) * width * height));
        CHECK(hipMemcpy(input_d, sample, sizeof(float) * width * height, hipMemcpyHostToDevice));

        const int
            BLOCK_DIM_X = 32,
            BLOCK_DIM_Y = 32;

        const dim3
            blockDim(BLOCK_DIM_X, BLOCK_DIM_Y),
            gridDim(divup(width, BLOCK_DIM_X), divup(height, BLOCK_DIM_Y));

        cudaCallbackKernel<<<
            gridDim,
            blockDim>>>(
            width,
            height,
            input_d,
            output_d);

        *result = (float *)malloc(sizeof(float) * width * height);
        CHECK(hipMemcpy(*result, output_d, sizeof(float) * width * height, hipMemcpyDeviceToHost));
        CHECK(hipFree(input_d));
        CHECK(hipFree(output_d));
    }
} // namespace v7
namespace v8 //cuda 预处理log到寄存器+使用更小的整型类型+使用更小的浮点类型+使用texure memory优化读入
{
    texture<float, 2> input_tex;

    __global__ void cudaCallbackKernel(
        const int width,
        const int height,
        float *__restrict__ output)
    {
        const int idy = blockIdx.y * blockDim.y + threadIdx.y;
        const int idx = blockIdx.x * blockDim.x + threadIdx.x;
        if (idy < height && idx < width)
        {
            signed char cnt[16] = {0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0};
            for (signed char offsety = -2; offsety <= 2; ++offsety)
            {
                const int py = idy + offsety;
                if (0 <= py && py < height)
                    for (signed char offsetx = -2; offsetx <= 2; ++offsetx)
                    {
                        const int px = idx + offsetx;
                        if (0 <= px && px < width)
                            ++cnt[(signed char)tex2D(input_tex, px, py)];
                    }
            }
            const float mylog[26] = {
                0.0,
                log(1.0),
                log(2.0),
                log(3.0),
                log(4.0),
                log(5.0),
                log(6.0),
                log(7.0),
                log(8.0),
                log(9.0),
                log(10.0),
                log(11.0),
                log(12.0),
                log(13.0),
                log(14.0),
                log(15.0),
                log(16.0),
                log(17.0),
                log(18.0),
                log(19.0),
                log(20.0),
                log(21.0),
                log(22.0),
                log(23.0),
                log(24.0),
                log(25.0)};

            const signed char n = neighborhood_size(idx, idy, width, height);
            double ans = mylog[n], inv_n = 1.0 / n;
            for (signed char i = 0; i < 16; ++i)
                ans -= mylog[cnt[i]] * inv_n * cnt[i];
            output[idy * width + idx] = ans;
        }
    }

    void cudaCallback(
        int width,
        int height,
        float *sample,
        float **result)
    {
        float *output_d;

        CHECK(hipMalloc((void **)&output_d, sizeof(float) * width * height));
        hipChannelFormatDesc channelDesc = hipCreateChannelDesc<float>();
        hipArray *cuArray;
        CHECK(hipMallocArray(&cuArray, &channelDesc, width, height));
        CHECK(hipMemcpyToArray(cuArray, 0, 0, sample, sizeof(float) * width * height, hipMemcpyHostToDevice));
        CHECK(hipBindTextureToArray(input_tex, cuArray));

        const int
            BLOCK_DIM_X = 32,
            BLOCK_DIM_Y = 32;

        const dim3
            blockDim(BLOCK_DIM_X, BLOCK_DIM_Y),
            gridDim(divup(width, BLOCK_DIM_X), divup(height, BLOCK_DIM_Y));

        cudaCallbackKernel<<<
            gridDim,
            blockDim>>>(
            width,
            height,
            output_d);

        *result = (float *)malloc(sizeof(float) * width * height);
        CHECK(hipMemcpy(*result, output_d, sizeof(float) * width * height, hipMemcpyDeviceToHost));
        CHECK(hipFreeArray(cuArray));
        CHECK(hipFree(output_d));
    }
} // namespace v8
namespace v9 //cuda 预处理log到寄存器+使用更小的整型类型+使用更小的浮点类型+使用shared memory优化读入
{
    template <
        int BLOCK_DIM_X,
        int BLOCK_DIM_Y>
    __global__ void cudaCallbackKernel(
        const int width,
        const int height,
        const float *__restrict__ input,
        float *__restrict__ output)
    {
        const int idy = blockIdx.y * (BLOCK_DIM_Y - 4) + threadIdx.y - 2;
        const int idx = blockIdx.x * (BLOCK_DIM_X - 4) + threadIdx.x - 2;
        __shared__ int input_s[BLOCK_DIM_Y][BLOCK_DIM_X | 1];

        input_s[threadIdx.y][threadIdx.x] = 0 <= idy && idy < height && 0 <= idx && idx < width ? input[idy * width + idx] : 16;

        __syncthreads();

        if (1 < threadIdx.y && threadIdx.y < BLOCK_DIM_Y - 2 &&
            1 < threadIdx.x && threadIdx.x < BLOCK_DIM_X - 2 &&
            idy < height && idx < width)
        {
            signed char cnt[17] = {0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0};
            for (signed char offsety = -2; offsety <= 2; ++offsety)
                for (signed char offsetx = -2; offsetx <= 2; ++offsetx)
                    ++cnt[input_s[threadIdx.y + offsety][threadIdx.x + offsetx]];

            const float mylog[26] = {
                0.0,
                log(1.0),
                log(2.0),
                log(3.0),
                log(4.0),
                log(5.0),
                log(6.0),
                log(7.0),
                log(8.0),
                log(9.0),
                log(10.0),
                log(11.0),
                log(12.0),
                log(13.0),
                log(14.0),
                log(15.0),
                log(16.0),
                log(17.0),
                log(18.0),
                log(19.0),
                log(20.0),
                log(21.0),
                log(22.0),
                log(23.0),
                log(24.0),
                log(25.0)};

            const signed char n = neighborhood_size(idx, idy, width, height);
            double ans = mylog[n], inv_n = 1.0 / n;
            for (signed char i = 0; i < 16; ++i)
                ans -= mylog[cnt[i]] * inv_n * cnt[i];
            output[idy * width + idx] = ans;
        }
    }

    void cudaCallback(
        int width,
        int height,
        float *sample,
        float **result)
    {
        float *input_d, *output_d;

        CHECK(hipMalloc((void **)&input_d, sizeof(float) * width * height));
        CHECK(hipMalloc((void **)&output_d, sizeof(float) * width * height));
        CHECK(hipMemcpy(input_d, sample, sizeof(float) * width * height, hipMemcpyHostToDevice));

        const int
            BLOCK_DIM_X = 32,
            BLOCK_DIM_Y = 32;

        const dim3
            blockDim(BLOCK_DIM_X, BLOCK_DIM_Y),
            gridDim(divup(width, BLOCK_DIM_X - 4), divup(height, BLOCK_DIM_Y - 4));

        cudaCallbackKernel<
            BLOCK_DIM_X,
            BLOCK_DIM_Y><<<
            gridDim,
            blockDim>>>(
            width,
            height,
            input_d,
            output_d);

        *result = (float *)malloc(sizeof(float) * width * height);
        CHECK(hipMemcpy(*result, output_d, sizeof(float) * width * height, hipMemcpyDeviceToHost));
        CHECK(hipFree(input_d));
        CHECK(hipFree(output_d));
    }
} // namespace v9
namespace v10 //openmp baseline
{
    void cudaCallback(
        int width,
        int height,
        float *sample,
        float **result)
    {
        *result = (float *)malloc(sizeof(float) * width * height);
#pragma omp parallel for
        for (int pos = 0; pos < width * height; ++pos)
        {
            const int
                idy = pos / width,
                idx = pos - idy * width;
            int cnt[16] = {0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0};
            for (int offsety = -2; offsety <= 2; ++offsety)
                for (int offsetx = -2; offsetx <= 2; ++offsetx)
                {
                    const int py = idy + offsety, px = idx + offsetx;
                    if (0 <= py && py < height && 0 <= px && px < width)
                        ++cnt[(int)sample[py * width + px]];
                }
            double n = neighborhood_size(idx, idy, width, height), ans = log(n);
            for (int i = 0; i < 16; ++i)
                if (cnt[i])
                    ans -= log((double)cnt[i]) * cnt[i] / n;
            (*result)[pos] = ans;
        }
    }
} // namespace v10
namespace v11 //openmp 预处理log到寄存器
{
    void cudaCallback(
        int width,
        int height,
        float *sample,
        float **result)
    {
        *result = (float *)malloc(sizeof(float) * width * height);
#pragma omp parallel for
        for (int pos = 0; pos < width * height; ++pos)
        {
            const int
                idy = pos / width,
                idx = pos - idy * width;
            int cnt[16] = {0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0};
            for (int offsety = -2; offsety <= 2; ++offsety)
                for (int offsetx = -2; offsetx <= 2; ++offsetx)
                {
                    const int py = idy + offsety, px = idx + offsetx;
                    if (0 <= py && py < height && 0 <= px && px < width)
                        ++cnt[(int)sample[py * width + px]];
                }
            const double mylog[26] = {
                0.0,
                log(1.0),
                log(2.0),
                log(3.0),
                log(4.0),
                log(5.0),
                log(6.0),
                log(7.0),
                log(8.0),
                log(9.0),
                log(10.0),
                log(11.0),
                log(12.0),
                log(13.0),
                log(14.0),
                log(15.0),
                log(16.0),
                log(17.0),
                log(18.0),
                log(19.0),
                log(20.0),
                log(21.0),
                log(22.0),
                log(23.0),
                log(24.0),
                log(25.0)};
            const int n = neighborhood_size(idx, idy, width, height);
            double ans = mylog[n], inv_n = 1.0 / n;
            for (int i = 0; i < 16; ++i)
                ans -= mylog[cnt[i]] * inv_n * cnt[i];
            (*result)[pos] = ans;
        }
    }
} // namespace v11
namespace v12 //openmp 预处理log到寄存器+使用更小的类型
{
    void cudaCallback(
        int width,
        int height,
        float *sample,
        float **result)
    {
        *result = (float *)malloc(sizeof(float) * width * height);
#pragma omp parallel for
        for (int pos = 0; pos < width * height; ++pos)
        {
            const int
                idy = pos / width,
                idx = pos - idy * width;
            signed char cnt[16] = {0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0};
            for (signed char offsety = -2; offsety <= 2; ++offsety)
                for (signed char offsetx = -2; offsetx <= 2; ++offsetx)
                {
                    const int py = idy + offsety, px = idx + offsetx;
                    if (0 <= py && py < height && 0 <= px && px < width)
                        ++cnt[(signed char)sample[py * width + px]];
                }
            const float mylog[26] = {
                0.0,
                log(1.0),
                log(2.0),
                log(3.0),
                log(4.0),
                log(5.0),
                log(6.0),
                log(7.0),
                log(8.0),
                log(9.0),
                log(10.0),
                log(11.0),
                log(12.0),
                log(13.0),
                log(14.0),
                log(15.0),
                log(16.0),
                log(17.0),
                log(18.0),
                log(19.0),
                log(20.0),
                log(21.0),
                log(22.0),
                log(23.0),
                log(24.0),
                log(25.0)};
            const signed char n = neighborhood_size(idx, idy, width, height);
            double ans = mylog[n], inv_n = 1.0 / n;
            for (signed char i = 0; i < 16; ++i)
                ans -= mylog[cnt[i]] * inv_n * cnt[i];
            (*result)[pos] = ans;
        }
    }
} // namespace v12
namespace v13 //openmp 预处理log到寄存器+使用更小的类型+预处理前缀和
{
    void cudaCallback(
        int width,
        int height,
        float *sample,
        float **result)
    {
        *result = (float *)malloc(sizeof(float) * width * height);
        int *sum[16];
#pragma omp parallel for
        for (int i = 0; i < 16; ++i)
        {
            int *p = (int *)malloc(sizeof(int) * (width + 5) * (height + 5));
            for (int pos = 0; pos < (width + 5) * (height + 5); ++pos)
            {
                const int idy = pos / (width + 5), idx = pos - idy * (width + 5);
                if (idy && idx)
                {
                    p[pos] = p[(idy - 1) * (width + 5) + idx] + p[idy * (width + 5) + idx - 1] - p[(idy - 1) * (width + 5) + (idx - 1)];
                    const int py = idy - 3, px = idx - 3;
                    if (0 <= py && py < height && 0 <= px && px < width && i == sample[py * width + px])
                        ++p[pos];
                }
                else
                    p[pos] = 0;
            }
            sum[i] = p;
        }
#pragma omp parallel for
        for (int pos = 0; pos < width * height; ++pos)
        {
            const int
                idy = pos / width,
                idx = pos - idy * width;
            const float mylog[26] = {
                0.0,
                log(1.0),
                log(2.0),
                log(3.0),
                log(4.0),
                log(5.0),
                log(6.0),
                log(7.0),
                log(8.0),
                log(9.0),
                log(10.0),
                log(11.0),
                log(12.0),
                log(13.0),
                log(14.0),
                log(15.0),
                log(16.0),
                log(17.0),
                log(18.0),
                log(19.0),
                log(20.0),
                log(21.0),
                log(22.0),
                log(23.0),
                log(24.0),
                log(25.0)};
            const signed char n = neighborhood_size(idx, idy, width, height);
            double ans = mylog[n], inv_n = 1.0 / n;
            for (signed char i = 0; i < 16; ++i)
            {
                const signed char cnti = sum[i][(idy + 5) * (width + 5) + idx + 5] - sum[i][(idy + 5) * (width + 5) + idx] - sum[i][idy * (width + 5) + idx + 5] + sum[i][idy * (width + 5) + idx];
                ans -= mylog[cnti] * inv_n * cnti;
            }
            (*result)[pos] = ans;
        }
        for (int i = 0; i < 16; ++i)
            free(sum[i]);
    }
} // namespace v13
namespace v14
{
    void cudaCallback(
        int width,
        int height,
        float *sample,
        float **result)
    {
        int num_threads = 0;
        hipGetDeviceCount(&num_threads);
        if (num_threads > height - 4)
            num_threads = height - 4;
        if (num_threads < 1)
            return v11::cudaCallback(width, height, sample, result);
        if (num_threads < 2 || width * height < 1e6)
            return v7::cudaCallback(width, height, sample, result);
        *result = (float *)malloc(sizeof(float) * width * height);
#pragma omp parallel num_threads(num_threads)
        {
            int thread_num = omp_get_thread_num(),
                thread_len = (height - 4) / num_threads,
                thread_beg = thread_len * thread_num + 2;
            if (thread_num == num_threads - 1)
                thread_len = height - thread_beg - 2;
            float *thread_result;
            hipSetDevice(thread_num);
            v9::cudaCallback(
                width,
                thread_len + 4,
                sample + width * (thread_beg - 2),
                &thread_result);
            memcpy(
                result + width * thread_beg,
                thread_result + width * 2,
                sizeof(float) * thread_len);
            if (thread_num == 0)
                memcpy(
                    result,
                    thread_result,
                    sizeof(float) * width * 2);
            if (thread_num == num_threads - 1)
                memcpy(
                    result + width * (height - 2),
                    thread_result + width * (thread_len - 2),
                    sizeof(float) * width * 2);
            free(thread_result);
        }
    }
} // namespace v14
void cudaCallback(
    int width,
    int height,
    float *sample,
    float **result)
{
    v14::cudaCallback(width, height, sample, result);
}
